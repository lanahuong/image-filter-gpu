#include <hip/hip_runtime.h>

/** Saturate one color component of the image
 * @param img the final image
 * @param rgb_sat a number that identify the component to saturate : red is 0, green is 1 and blue is 2
 * @param size the number of pixels that compose an image
 */
__global__
void kernel_sat1(unsigned int* img, int rgb_sat, unsigned size) {
    // Compute index of thread
    int g_block_idx = blockIdx.x + blockIdx.y * gridDim.x;
    int th_block = blockDim.x * blockDim.y;
    int g_idx = g_block_idx * th_block + threadIdx.x + threadIdx.y * blockDim.x;

    // Compute index of pixel to alter
    int k = g_idx * 3;

    // Correct rgb_sat to be in {0,1,2}
    rgb_sat = rgb_sat % 3;

    // Saturate the componant rgb_sat
    if (g_idx<size) {
        img[k + rgb_sat] = 0xFF;
    }
}

/** Turn the image to greyscale
 * @param img the image to modify and final image
 * @param size the number of pixels that compose an image
 */
__global__
void kernel_grey(unsigned int* img, unsigned size) {
    // Compute index of thread
    int g_block_idx = blockIdx.x + blockIdx.y * gridDim.x;
    int th_block = blockDim.x * blockDim.y;
    int g_idx = g_block_idx * th_block + threadIdx.x + threadIdx.y * blockDim.x;

    // Compute index of pixel to alter
    int k = g_idx * 3;

    // Compute grey color and assign it to all componant
    if (g_idx<size) {
        int grey = img[k+0]*0.299 + img[k+1]*0.587 + img[k+2]*0.114;
        img[k] = grey;
        img[k + 1] = grey;
        img[k + 2] = grey;
    }
}

/** Flip the image horizontally
 * @param new_img the final image
 * @param img a copy of the image to modify
 * @param width the pixel width of the image
 * @param height the pixel height of the image
 */
__global__
void kernel_hmirror(unsigned int* new_img, unsigned int* img, unsigned width, unsigned height) {
    unsigned size = width * height;

    // Compute index of thread
    int g_block_idx = blockIdx.x + blockIdx.y * gridDim.x;
    int th_block = blockDim.x * blockDim.y;
    int g_idx = g_block_idx * th_block + threadIdx.x + threadIdx.y * blockDim.x;

    // Compute index of pixel to alter
    int k = g_idx * 3;

    // Compute the index of the last pixel in the line
    int i = ((g_idx / width) + 1) * width * 3 - 3;
    // Compute the column of the pixel to alter
    int j = (g_idx % width) * 3;

    // Assign each pixel the color of the opposite pixel on the line
    if (g_idx<size) {
        new_img[k] = img[i-j];
        new_img[k + 1] = img[i-j+1];
        new_img[k + 2] = img[i-j+2];
    }
}

/** Flip the image vertically
 * @param new_img the final image
 * @param img a copy of the image to modify
 * @param width the pixel width of the image
 * @param height the pixel height of the image
 */
__global__
void kernel_vmirror(unsigned int* new_img, unsigned int* img, unsigned width, unsigned height) {
    unsigned size = width * height;

    // Compute index of thread
    int g_block_idx = blockIdx.x + blockIdx.y * gridDim.x;
    int th_block = blockDim.x * blockDim.y;
    int g_idx = g_block_idx * th_block + threadIdx.x + threadIdx.y * blockDim.x;

    // Compute index of pixel to alter
    int k = g_idx * 3;

    // Compute the index of the last pixel in the column
    int kk = ((g_idx % width) + (height - 1) * width) * 3;
    // Compute the offset of the row of the pixel to alter
    int i = (g_idx / width) * 3 * width;

    // Assign each pixel the color of the opposite pixel on the line
    if (g_idx<size) {
        new_img[k] = img[kk - i];
        new_img[k + 1] = img[kk - i + 1];
        new_img[k + 2] = img[kk - i + 2];
    }
}


/** Simple blur using direct neighbors (up, down, left and right)
 * @param new_img the final image
 * @param img a copy of the image to modify
 * @param width the pixel width of the image
 * @param height the pixel height of the image
 */
__global__
void kernel_blur(unsigned int* new_img, unsigned int* img, unsigned width, unsigned height) {
    unsigned size = width * height;

    // Compute index of thread
    int g_block_idx = blockIdx.x + blockIdx.y * gridDim.x;
    int th_block = blockDim.x * blockDim.y;
    int g_idx = g_block_idx * th_block + threadIdx.x + threadIdx.y * blockDim.x;

    // Compute index of pixel to alter
    int k = g_idx * 3;

    // Compute the row and column of the pixel to alter
    int i = g_idx / width;
    int j = g_idx % width;

    if (g_idx<size && j > 0 && j < width - 1 && i > 0 && i < height - 1) {
        unsigned v[4] = {
                         ((i-1)*width+j)*3,
                         (i*width+j-1)*3,
                         (i*width+j+1)*3,
                         ((i+1)*width+j)*3,
                        };
        new_img[k] = (img[k] + img[v[0]] + img[v[1]] + img[v[2]] + img[v[3]]) / 5;
        new_img[k + 1] = (img[k + 1] + img[v[0] + 1] + img[v[1] + 1] + img[v[2] + 1] + img[v[3] + 1]) / 5;
        new_img[k + 2] = (img[k + 2] + img[v[0] + 2] + img[v[1] + 2] + img[v[2] + 2] + img[v[3] + 2]) / 5;
    }
}

/** Compute the convolution of an image with a kernel (matrix)
 * @param new_img the final image
 * @param img a copy of the image to modify
 * @param width the pixel width of the image
 * @param height the pixel height of the image
 * @param kernel the kernel (matrix) to use for the convolution
 * @param kernel_size the side dimension of the kernel (if size is 3 then the kernel has 9 coefficient), it should be an odd number
 */
__global__
void kernel_convolution_rgb(unsigned int* new_img, unsigned int* img, unsigned width, unsigned height, float* kernel, unsigned int kernel_size) {
    unsigned size = width * height;

    // Compute index of thread
    int g_block_idx = blockIdx.x + blockIdx.y * gridDim.x;
    int th_block = blockDim.x * blockDim.y;
    int b_idx = threadIdx.x + threadIdx.y * blockDim.x;
    int g_idx = g_block_idx * th_block + b_idx;

    // Compute index of pixel to alter
    int k = g_idx * 3;

    // Compute the row and column of the pixel to alter
    int i = g_idx / width;
    int j = g_idx % width;

    int kernel_center = (kernel_size - 1) / 2;
    // Efficiently copy the kernel in the block shared memory
    int ks2 = kernel_size*kernel_size;
    extern __shared__ float ker[];
    if (b_idx<ks2)
        ker[b_idx] = kernel[b_idx];
    __syncthreads();

    // Compute convolution for each color
    if (g_idx<size && j >= kernel_center && j < (width - kernel_center) && i >= kernel_center && i < (height - kernel_center)) {
        int r = 0;
        int g = 0;
        int b = 0;
        for (int ki = 0; ki<kernel_size; ki++) {
            for (int kj = 0; kj<kernel_size; kj++) {
                int ii = i + ki - kernel_center;
                int jj = j + kj - kernel_center;
                int kk = (ii*width + jj)*3;
                r += img[kk] * ker[ki * kernel_size + kj];
                g += img[kk + 1] * ker[ki * kernel_size + kj];
                b += img[kk + 2] * ker[ki * kernel_size + kj];
            }
        }
        new_img[k] = r;
        new_img[k + 1] = g;
        new_img[k + 2] = b;
    }
}

/** Simple blur using direct neighbors (up, down, left and right) with a convolution
 * @param d_img the final image on GPU
 * @param d_img_tmp a copy of the image to modify on GPU
 * @param width the pixel width of the image
 * @param height the pixel height of the image
 * @param kernel blockSize the block dimensions for launching the kernel
 * @param kernel gridSize the grid dimensions for launching the kernel
 */
void run_blur_v2(unsigned int* d_img, unsigned int* d_img_tmp, unsigned width, unsigned height, dim3 blockSize, dim3 gridSize) {
  // Create the kernel and send it to the GPU
  float kernel[9] = {0.f, 0.2f, 0.f, 0.2f, 0.2f, 0.2f, 0.f, 0.2f, 0.f};
  float *d_kernel;
  hipMalloc((void **) &d_kernel, 9*sizeof(float));
  hipMemcpy(d_kernel, &kernel, 9*sizeof(float), hipMemcpyHostToDevice);

  // Lauch the kernel
  kernel_convolution_rgb<<<gridSize,blockSize, 9*sizeof(float)>>>(d_img, d_img_tmp, width, height, d_kernel, 3);

  hipFree(d_kernel);
}

/** Blur an image with an average blur of given radius
 * @param d_img the final image on GPU
 * @param d_img_tmp a copy of the image to modify on GPU
 * @param width the pixel width of the image
 * @param height the pixel height of the image
 * @param blockSize the block dimensions for launching the kernel
 * @param gridSize the grid dimensions for launching the kernel
 * @param r the radius of the blur kernel
 */
void run_blur(unsigned int* d_img, unsigned int* d_img_tmp, unsigned width, unsigned height, dim3 blockSize, dim3 gridSize, int r) {
  int k_size = 2 * r + 1;
  int k_size2 = k_size * k_size;
  int k_alloc = sizeof(float) * k_size2;
  // Create the kernel and send it to the GPU
  float *kernel = (float*) malloc(k_alloc);
  float val = 1.f/(float)k_size2;
  for (int i = 0; i < k_size2; i++) {
      kernel[i] = val;
  }
  float *d_kernel;
  hipMalloc((void **) &d_kernel, k_alloc);
  hipMemcpy(d_kernel, kernel, k_alloc, hipMemcpyHostToDevice);

  // Lauch the kernel
  kernel_convolution_rgb<<<gridSize,blockSize, k_alloc>>>(d_img, d_img_tmp, width, height, d_kernel, k_size);

  free(kernel);
  hipFree(d_kernel);
}

/** Compute the SOBEL filter on a greyscale image
 * @param new_img the final image
 * @param img the image to modify
 * @param width the pixel width of the image
 * @param height the pixel height of the image
 * @param kernel the sobel kernels (2 in 1)
 */
__global__
void kernel_sobel(unsigned int* new_img, unsigned int* img, unsigned width, unsigned height, int* kernels) {
    unsigned size = width * height;

    // Compute index of thread
    int g_block_idx = blockIdx.x + blockIdx.y * gridDim.x;
    int th_block = blockDim.x * blockDim.y;
    int b_idx = threadIdx.x + threadIdx.y * blockDim.x;
    int g_idx = g_block_idx * th_block + b_idx;

    // Compute index of pixel to alter
    int k = g_idx * 3;

    // Compute the row and column of the pixel to alter
    int i = g_idx / width;
    int j = g_idx % width;

    // Efficiently copy the kernel in the block shared memory
    __shared__ float ker[9];
    if (b_idx<9)
        ker[b_idx] = kernels[b_idx];
    __syncthreads();

    if (g_idx<size && j >= 1 && j < (width - 1) && i >= 1 && i < (height - 1)) {
        int gx = 0;
        int gy = 0;
        for (int ki = 0; ki<3; ki++) {
            for (int kj = 0; kj<3; kj++) {
                int ii = i + ki - 1;
                int jj = j + kj - 1;
                int kk = (ii*width + jj)*3;
                gx += img[kk] * ker[ki * 3 + kj];
                gy += img[kk] * ker[kj * 3 + ki];
            }
        }
        int val = sqrt((float) (gx*gx + gy*gy));
        new_img[k] = val;
        new_img[k + 1] = val;
        new_img[k + 2] = val;
    }
}

/** Run the SOBEL edge detection filter
 * @param img the image on CPU
 * @param d_img the final image on GPU
 * @param d_img_tmp a copy of the image to modify on GPU
 * @param width the pixel width of the image
 * @param height the pixel height of the image
 * @param blockSize the block dimensions for launching the kernel
 * @param gridSize the grid dimensions for launching the kernel
 */
void run_sobel(unsigned int* img, unsigned int* d_img, unsigned int* d_img_tmp, unsigned width, unsigned height, dim3 blockSize, dim3 gridSize) {
  unsigned image_size = width*height;
  unsigned alloc_size = sizeof(unsigned int) * image_size * 3;

  kernel_grey<<<gridSize,blockSize>>>(d_img, image_size);
  hipMemcpy(img, d_img, alloc_size, hipMemcpyDeviceToHost);
  hipMemcpy(d_img_tmp, img, alloc_size, hipMemcpyHostToDevice);

  // Create the kernel and send it to the GPU
  int kernel[9] = {-1, 0, 1, -2, 0, 2, -1, 0, 1};
  //float kernel[9] = {0.f, 0.2f, 0.f, 0.2f, 0.2f, 0.2f, 0.f, 0.2f, 0.f};
  int *d_kernel;
  hipMalloc((void **) &d_kernel, 9*sizeof(int));
  hipMemcpy(d_kernel, &kernel, 9*sizeof(int), hipMemcpyHostToDevice);

  // Lauch the kernel
  kernel_sobel<<<gridSize,blockSize>>>(d_img, d_img_tmp, width, height, d_kernel);

  hipFree(d_kernel);
}

/** Compute the reduction of an image to a forth of it's size
 * @param new_img the final image it has the size of the reduced image
 * @param img the image to modify
 * @param width the pixel width of the original image
 * @param height the pixel height of the original image
 */
__global__
void kernel_reduction(unsigned int* new_img, unsigned int* img, unsigned width, unsigned height) {
    unsigned size = width * height * 0.25;

    // Compute index of thread
    int g_block_idx = blockIdx.x + blockIdx.y * gridDim.x;
    int th_block = blockDim.x * blockDim.y;
    int g_idx = g_block_idx * th_block + threadIdx.x + threadIdx.y * blockDim.x;

    // Compute index of pixel to alter
    int k = g_idx * 3;

    // Compute the row and column of the pixel of the new image to set
    int i = 2 * g_idx / width;
    int j = g_idx % (width/2);

    if (g_idx<size) {
        // For each pixel we compute the average of a square of 4
        int ii = 2 * i;
        int jj = 2 * j;

        int kk = (ii * width + jj) * 3;
        int kk_next = ((ii+1) * width + jj) * 3;

        new_img[k] = (img[kk] + img[kk+3] + img[kk_next] + img[kk_next+3]) / 4;
        new_img[k + 1] = (img[kk + 1] + img[kk + 4] + img[kk_next + 1] + img[kk_next + 4]) / 4 ;
        new_img[k + 2] = (img[kk + 2] + img[kk + 5] + img[kk_next + 2] + img[kk_next + 5]) / 4;
    }
}

/** Copy a small image in one corner of a large one
 * @param new_img a pointer to the starting pixel where the small image shoud be copied
 * @param img the small image to copy back in a corner of the frame
 * @param width the pixel width of the small image
 * @param height the pixel height of the small image
 */
__global__
void kernel_recompose(unsigned int* new_img, unsigned int* img, unsigned width, unsigned height) {
    unsigned size = width * height;

    // Compute index of thread
    int g_block_idx = blockIdx.x + blockIdx.y * gridDim.x;
    int th_block = blockDim.x * blockDim.y;
    int b_idx = threadIdx.x + threadIdx.y * blockDim.x;
    int g_idx = g_block_idx * th_block + b_idx;

    // Compute index of pixel to copy
    int k = g_idx * 3;

    // Compute the row and column of the pixel to copy
    int i = g_idx / width;
    int j = g_idx % width;

    // Compute index of pixel to set
    int kk = (i * width * 2 + j) * 3;

    if (g_idx<size) {
        new_img[kk] = img[k];
        new_img[kk + 1] = img[k + 1];
        new_img[kk + 2] = img[k + 2];
    }
}

/** Apply a pop-art filter on the image
 * @param img the image on CPU
 * @param d_img the final image on GPU
 * @param d_img_tmp a copy of the image to modify on GPU
 * @param width the pixel width of the image
 * @param height the pixel height of the image
 */
void run_popart(unsigned int* img, unsigned int* d_img, unsigned int* d_img_tmp, unsigned width, unsigned height) {
  unsigned image_size_small = width * height / 4;
  unsigned alloc_size_small = sizeof(unsigned int) * image_size_small * 3;

  dim3 blockSize(32,32);
  dim3 gridSize(0,0);
  gridSize.x = width / 64 +1;
  gridSize.y = height / 64 +1;

  unsigned int *img_small = (unsigned int*) malloc(alloc_size_small);
  unsigned int *d_img_small;

  hipMalloc((void **) &d_img_small, alloc_size_small);

  kernel_reduction<<<gridSize,blockSize>>>(d_img_small, d_img, width, height);

  hipMemcpy(img_small, d_img_small, alloc_size_small, hipMemcpyDeviceToHost);

  unsigned int nstreams = 4;
  hipStream_t stream[nstreams];
  for (int i = 0; i<nstreams; i++) {
      hipStreamCreate(&stream[i]);
  }

  int offset_final[4];
  offset_final[0] = 0;
  offset_final[1] = 3 * width / 2;
  offset_final[2] = image_size_small * 6;
  offset_final[3] = offset_final[2] + offset_final[1];

  for (int i = 0; i<nstreams; i++) {
    int offset = i * image_size_small * 3;
    hipMemcpyAsync(d_img_tmp+offset, img_small, alloc_size_small, hipMemcpyHostToDevice, stream[i]);
    if (i == 0) {
      kernel_grey<<<gridSize,blockSize,0,stream[i]>>>(d_img_tmp, image_size_small);
    } else {
        kernel_sat1<<<gridSize,blockSize,0,stream[i]>>>(d_img_tmp+offset, i, image_size_small);
    }
    kernel_recompose<<<gridSize,blockSize,0,stream[i]>>>(d_img+offset_final[i], d_img_tmp+offset, width/2, height/2);
  }

  for (int i = 0; i<nstreams; i++) {
    int offset = i * image_size_small * 3;
    hipMemcpyAsync(img+offset, d_img+offset, alloc_size_small, hipMemcpyDeviceToHost, stream[i]);
  }

  hipDeviceSynchronize();

  free(img_small);
  hipFree(d_img_small);
}

/** Negate the image
 * @param img the image to modify and final image
 * @param size the number of pixels that compose an image
 */
__global__
void kernel_negative(unsigned int* img, unsigned size) {
    // Compute index of thread
    int g_block_idx = blockIdx.x + blockIdx.y * gridDim.x;
    int th_block = blockDim.x * blockDim.y;
    int g_idx = g_block_idx * th_block + threadIdx.x + threadIdx.y * blockDim.x;

    // Compute index of pixel to alter
    int k = g_idx * 3;

    // Compute grey color and assign it to all componant
    if (g_idx<size) {
        img[k] = 255 - img[k];
        img[k + 1] = 255 - img[k + 1];
        img[k + 2] = 255 - img[k + 2];
    }
}

/** Turn the image black and white depending on a given threashold
 * @param img the image to modify and final image
 * @param size the number of pixels that compose an image
 * @param threashold the threashold above which the pixel is black else it's black
 */
__global__
void kernel_binary(unsigned int* img, unsigned size, int threashold) {
    // Compute index of thread
    int g_block_idx = blockIdx.x + blockIdx.y * gridDim.x;
    int th_block = blockDim.x * blockDim.y;
    int g_idx = g_block_idx * th_block + threadIdx.x + threadIdx.y * blockDim.x;

    // Compute index of pixel to alter
    int k = g_idx * 3;

    // Compute grey color and assign it to all componant
    if (g_idx<size) {
        int color = ((img[k+0]*0.299 + img[k+1]*0.587 + img[k+2]*0.114) >= threashold) * 255;
        img[k] = color;
        img[k + 1] = color;
        img[k + 2] = color;
    }
}
